
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ const char *STR = "HELLO WORLD!";
const char STR_LENGTH = 12;

__global__ void hello()
{
	printf("%c from thread %d\n", STR[threadIdx.x % STR_LENGTH], threadIdx.x);
}

int main(void)
{
	int num_threads = STR_LENGTH;
	int num_blocks = 1;
	hello<<<num_blocks,num_threads>>>();
	hipDeviceSynchronize();

	return 0;
}
