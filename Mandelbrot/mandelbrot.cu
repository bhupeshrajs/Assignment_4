
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

/* 
 *  Used in Serial Implementation of the mandelbrot 
 */
int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i) {

        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re*z_re - z_im*z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}


/* 
 *  Mandelbrot Serial Function 
 */
void mandelbrotSerial(
    float x0, float y0, float x1, float y1,
    int width, int height,
    int startRow, int totalRows,
    int maxIterations,
    int output[])
{
    float dx = (x1 - x0) / width;
    float dy = (y1 - y0) / height;

    int endRow = startRow + totalRows;

    for (int j = startRow; j < endRow; j++) {
        for (int i = 0; i < width; ++i) {
            float x = x0 + i * dx;
            float y = y0 + j * dy;

            int index = (j * width + i);
            output[index] = mandel(x, y, maxIterations);
        }
    }
}

/*
 *  Used to check the output of serial and CUDA Executions.   
 */
bool verifyResult (int *gold, int *result, int width, int height) {

    int i, j;

    for (i = 0; i < height; i++) {
        for (j = 0; j < width; j++) {
            if (gold[i * width + j] != result[i * width + j]) {
                printf ("Mismatch : [%d][%d], Expected : %d, Actual : %d\n",
                            i, j, gold[i * width + j], result[i * width + j]);
                continue;
            }
        }
    }

    return 1;
}

__global__ void mandelbrotCUDA(
                    float *d_x0, float *d_y0, float *d_x1, float *d_y1,
                    int *d_width, int *d_height,
                    int *d_maxIterations,
                    int *d_output_cuda ) {
    
    int row = blockIdx.y * blockDim.y + threadIdx.y; // HEIGHT
    int col = blockIdx.x * blockDim.x + threadIdx.x; // WIDTH
    
    if( row == 0 && col == 800 ) {
        printf("\n block dimensions are : (%d,%d)",blockDim.x,blockDim.y) ;
        printf("\n block Ids are : (%d,%d)",blockIdx.x,blockIdx.y) ;
        printf("\n thread Ids are : (%d,%d)",threadIdx.x,threadIdx.y) ;
        printf("\n x0, x1, y0, y1 : %f, %f, %f, %f",*d_x0,*d_x1,*d_y0,*d_y1);
        printf("\n height and width : %d, %d",*d_height,*d_width);
    }
    
    int index = row * (*d_width) + col;
    
    if( col >= (*d_width) ) return;
    if( row >= (*d_height) ) return;
    if( index >= ( (*d_height)*(*d_width) ) ) return;
    
    float dx = ( (*d_x1) - (*d_x0) ) / (*d_width);
    float dy = ( (*d_y1) - (*d_y0) ) / (*d_height);
    
    float c_re = (*d_x0) + col * dx;
    float c_im = (*d_y0) + row * dy;
    
    float z_re = c_re;
    float z_im = c_im;
    
    int i;
    for ( i = 0 ; i < *d_maxIterations ; i++ ) {
    
        if( z_re * z_re + z_im * z_im > 4.f ) 
            break;
            
        float new_re = z_re*z_re - z_im*z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    d_output_cuda[index] = i;
    
}

int main(int argc, char *argv[])
{

    
    if(argc < 2) {
        printf("\n Give number of threads per block dimension value.");
        return 1;
    }
    
    int thread_dimension = atoi(argv[1]);
    
    
    /* Height , width of the image */
    const int width = 1200;
    const int height = 800;
    
    /* Max iterations to do */
    const int maxIterations = 256;
    
    /* The value of x0,x1,y0,y1 */
    float x0 = -2;
    float x1 = 1;
    float y0 = -1;
    float y1 = 1;


    int* output_serial = (int*)malloc(width*height*sizeof(int));
    int* output_cuda = (int*)malloc(width*height*sizeof(int));
    
    int *d_output_cuda;
    float *d_x0;
    float *d_y0;
    float *d_x1;
    float *d_y1;
    int *d_width, *d_height;
    int *d_maxIterations;
    
    hipMalloc((void **)&d_output_cuda, sizeof(int)*width*height);
    hipMalloc((void **)&d_x0, sizeof(float));
    hipMalloc((void **)&d_x1, sizeof(float));
    hipMalloc((void **)&d_y0, sizeof(float));
    hipMalloc((void **)&d_y1, sizeof(float));
    hipMalloc((void **)&d_width, sizeof(int));
    hipMalloc((void **)&d_height, sizeof(int));
    hipMalloc((void **)&d_maxIterations, sizeof(int));
    
    hipMemcpy(d_output_cuda, output_cuda, sizeof(int)*width*height, hipMemcpyHostToDevice);
    hipMemcpy(d_x0, &x0, sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_x1, &x1, sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_y0, &y0, sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_y1, &y1, sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(d_width, &width, sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(d_height, &height, sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(d_maxIterations, &maxIterations, sizeof(int) , hipMemcpyHostToDevice);
    
    dim3 block_size(thread_dimension, thread_dimension);
    dim3 grid_size( ceil(width / block_size.x), ceil(height / block_size.y));
    
    mandelbrotCUDA<<<grid_size,block_size>>>(d_x0,d_y0,d_x1,d_y1,d_width,d_height,d_maxIterations,d_output_cuda);
    
    hipMemcpy(output_cuda, d_output_cuda, sizeof(int)*width*height, hipMemcpyDeviceToHost);
    
    mandelbrotSerial(x0, y0, x1, y1, width, height, 0, height, maxIterations, output_serial);
    
    hipFree(d_output_cuda);
    hipFree(d_x0);
    hipFree(d_x1);
    hipFree(d_y0);
    hipFree(d_y1);
    hipFree(d_width);
    hipFree(d_height);
    hipFree(d_maxIterations);

    if (! verifyResult (output_serial, output_cuda, width, height)) {
        printf ("Error : Output from threads does not match serial output\n");
        return 1;
    }
    else {
        printf("\n\"The output from the CUDA matches the serial output\"\n\n");
    }


    return 0;
}