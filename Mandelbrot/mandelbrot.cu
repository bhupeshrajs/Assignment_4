
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
/* 
 *  Used in Serial Implementation of the mandelbrot 
 */
int mandel(double c_re, double c_im, int count)
{
    double z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i) {

        if (z_re * z_re + z_im * z_im > 4.0)
            break;

        double new_re = z_re*z_re - z_im*z_im;
        double new_im = 2.0 * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}


/* 
 *  Mandelbrot Serial Function 
 */
void mandelbrotSerial(
    double x0, double y0, double x1, double y1,
    int width, int height,
    int startRow, int totalRows,
    int maxIterations,
    int output[])
{
    double dx = (x1 - x0) / width;
    double dy = (y1 - y0) / height;

    int endRow = startRow + totalRows;

    for (int j = startRow; j < endRow; j++) {
        for (int i = 0; i < width; ++i) {
            double x = x0 + i * dx;
            double y = y0 + j * dy;

            int index = (j * width + i);
            output[index] = mandel(x, y, maxIterations);
        }
    }
}

/*
 *  Used to check the output of serial and CUDA Executions.   
 */
bool verifyResult (int *gold, int *result, int width, int height) {

    int i, j;

    for (i = 0; i < height; i++) {
        for (j = 0; j < width; j++) {
            if (gold[i * width + j] != result[i * width + j]) {
                printf ("Mismatch : [%d][%d], Expected : %d, Actual : %d\n",
                            i, j, gold[i * width + j], result[i * width + j]);
                continue;
            }
        }
    }

    return 1;
}

__global__ void mandelbrotCUDA(
                    double *d_x0, double *d_y0, double *d_x1, double *d_y1,
                    int *d_width, int *d_height,
                    int *d_maxIterations,
                    int *d_output_cuda ) {
    
    int row = blockIdx.y * blockDim.y + threadIdx.y; // HEIGHT
    int col = blockIdx.x * blockDim.x + threadIdx.x; // WIDTH
    
    if( row == 0 && col == 800 ) {
        printf("\n block dimensions are : (%d,%d)",blockDim.x,blockDim.y) ;
        printf("\n block Ids are : (%d,%d)",blockIdx.x,blockIdx.y) ;
        printf("\n thread Ids are : (%d,%d)",threadIdx.x,threadIdx.y) ;
        printf("\n x0, x1, y0, y1 : %f, %f, %f, %f",*d_x0,*d_x1,*d_y0,*d_y1);
        printf("\n height and width : %d, %d",*d_height,*d_width);
    }
    
    int index = (row * (*d_width)) + col;
    
    if( col >= (*d_width) ) return;
    if( row >= (*d_height) ) return;
    if( index >= ( (*d_height)*(*d_width) ) ) return;
    
    double dx = ( (*d_x1) - (*d_x0) ) / (*d_width);
    double dy = ( (*d_y1) - (*d_y0) ) / (*d_height);
    
    double c_re = (*d_x0) + col * dx;
    double c_im = (*d_y0) + row * dy;
    
    double z_re = c_re;
    double z_im = c_im;
    
    int i = 0;
    for ( i = 0 ; i < *d_maxIterations ; ++i ) {
    
        if( z_re * z_re + z_im * z_im > 4.0 ) 
            break;
            
        double new_re = z_re*z_re - z_im*z_im;
        double new_im = 2.0 * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    d_output_cuda[index] = i;
    
}

int main(int argc, char *argv[])
{

    
    if(argc < 2) {
        printf("\n Give number of threads per block dimension value.");
        return 1;
    }
    
    int thread_dimension = atoi(argv[1]);
    
    
    /* Height , width of the image */
    const int width = 1200;
    const int height = 800;
    
    /* Max iterations to do */
    const int maxIterations = 256;
    
    /* The value of x0,x1,y0,y1 */
    double x0 = -2;
    double x1 = 1;
    double y0 = -1;
    double y1 = 1;


    int* output_serial = (int*)malloc(width*height*sizeof(int));
    int* output_cuda = (int*)malloc(width*height*sizeof(int));
    
    int *d_output_cuda;
    double *d_x0;
    double *d_y0;
    double *d_x1;
    double *d_y1;
    int *d_width, *d_height;
    int *d_maxIterations;
    
    hipMalloc((void **)&d_output_cuda, sizeof(int)*width*height);
    hipMalloc((void **)&d_x0, sizeof(double));
    hipMalloc((void **)&d_x1, sizeof(double));
    hipMalloc((void **)&d_y0, sizeof(double));
    hipMalloc((void **)&d_y1, sizeof(double));
    hipMalloc((void **)&d_width, sizeof(int));
    hipMalloc((void **)&d_height, sizeof(int));
    hipMalloc((void **)&d_maxIterations, sizeof(int));
    
    hipMemcpy(d_output_cuda, output_cuda, sizeof(int)*width*height, hipMemcpyHostToDevice);
    hipMemcpy(d_x0, &x0, sizeof(double) , hipMemcpyHostToDevice);
    hipMemcpy(d_x1, &x1, sizeof(double) , hipMemcpyHostToDevice);
    hipMemcpy(d_y0, &y0, sizeof(double) , hipMemcpyHostToDevice);
    hipMemcpy(d_y1, &y1, sizeof(double) , hipMemcpyHostToDevice);
    hipMemcpy(d_width, &width, sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(d_height, &height, sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(d_maxIterations, &maxIterations, sizeof(int) , hipMemcpyHostToDevice);
    
    dim3 block_size(thread_dimension, thread_dimension);
    dim3 grid_size( ceil(width / block_size.x), ceil(height / block_size.y));
    
    mandelbrotCUDA<<<grid_size,block_size>>>(d_x0,d_y0,d_x1,d_y1,d_width,d_height,d_maxIterations,d_output_cuda);
    
    hipMemcpy(output_cuda, d_output_cuda, sizeof(int)*width*height, hipMemcpyDeviceToHost);
    
    mandelbrotSerial(x0, y0, x1, y1, width, height, 0, height, maxIterations, output_serial);
    
    hipFree(d_output_cuda);
    hipFree(d_x0);
    hipFree(d_x1);
    hipFree(d_y0);
    hipFree(d_y1);
    hipFree(d_width);
    hipFree(d_height);
    hipFree(d_maxIterations);

    if (! verifyResult (output_serial, output_cuda, width, height)) {
        printf ("Error : Output from threads does not match serial output\n");
        return 1;
    }
    else {
        printf("\n\"The output from the CUDA matches the serial output\"\n\n");
    }


    return 0;
}